
#include <hip/hip_runtime.h>
#include <iostream>     // cout
#include <stdint.h>     // Standardized integers
#include <unistd.h>
#include <math.h>       // fabs


const uint16_t BLK_SIZE = 3;
const float EPSILON = 0.0001;   // For sanity check: float == float

void sanityCheck();
bool resultCheck(float *arr1, float *arr2);
void printArray(float *arr);

__global__ void matDotVec(float **arr, uint16_t size);

int main(int argc, char **argv){
    sanityCheck();
    return 0;
}



void sanityCheck()
{
    uint16_t n = BLK_SIZE;
    uint32_t bytes = n * sizeof(float);
    uint32_t ptr_bytes = n * sizeof(float*);        // Just being safe
    uint32_t bytes_sqrd = bytes * bytes;

    float **h_matrix = (float**)malloc(ptr_bytes);   // To store "Matrix A"
    // float **d_matrix1 = (float*)malloc(ptr_bytes);  // To store the data from host
    // float **d_matrix2 = (float*)malloc(ptr_bytes);  // To store the multiplication that happens
    float *h_vec = (float*)malloc(bytes);           // To store "Vector B" and result
    // float *d_vec = (float*)malloc(bytes);           // To store the data from host and result
    float *expected = (float*)malloc(bytes);        // Sanity check array

    float **d_matrix1;      // To store the data from host
    float **d_matrix2;      // To store the multiplication that happens
    float *d_vec;           // To store the data from host and result

    // Allocate memory
    for (uint16_t i = 0; i < n; ++i){
        h_matrix[i] = (float*)malloc(bytes);
        // d_matrix1[i] = (float*)malloc(bytes);
        // d_matrix2[i] = (float*)malloc(bytes);
        // memset(d_matrix1[i], 0.0f, bytes);
        // memset(d_matrix2[i], 0.0f, bytes);
    }

    float incr = 0.0;
    for (uint16_t i = 0; i < n; ++i){
        for (uint16_t j = 0; j < n; ++j){
            h_matrix[i][j] = ++incr;
        }
    }

    // "Vector B"
    h_vec[0] = 10.0f;
    h_vec[1] = 11.0f;
    h_vec[2] = 12.0f;
    
    // Expected values out of matrixCheck()
    expected[0] = 68.0f;
    expected[1] = 167.0f;
    expected[2] = 266.0f;
    

    // Allocate CUDA memory and copy data to device
    hipMalloc((float**)&d_matrix1, bytes_sqrd);
    hipMalloc((float**)&d_matrix2, bytes_sqrd);
    hipMalloc((float**)&d_vec, bytes);
    hipMemcpy(d_matrix1, h_matrix, bytes_sqrd, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, expected, bytes, hipMemcpyHostToDevice);

    // Call CUDA function here...
    matDotVec<<<1, n>>>(d_matrix1, n);

    // Copy result back to host & free memory
    hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_matrix1);
    hipFree(d_matrix2);

    if (resultCheck(expected, h_vec)){
        std::cout << "Sanity check PASSED!" << std::endl;
    }
    else{
        std::cout << "Sanity check FAILED!" << std::endl;
        
    }
    printArray(h_vec);

    // Free up device memory allocations
    free(h_matrix);
    free(h_vec);
    free(expected);
}



bool resultCheck(float *arr1, float *arr2){
    for (uint16_t i = 0; i < BLK_SIZE; ++i){
        if (fabs(arr1[i] - arr2[i]) > EPSILON){            
            return false;
        }
    }
    return true;
}




void printArray(float *arr){
    for (uint16_t i = 0; i < BLK_SIZE; ++i){
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}


__global__ void matDotVec(float **arr, uint16_t size){
    
}