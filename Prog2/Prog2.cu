#include "hip/hip_runtime.h"
#include <iostream>     // cout
#include <stdint.h>     // Standardized integers
#include <unistd.h>


const uint16_t BLK_SIZE = 3;

void sanityCheck();
bool arrayCheck(const float *a1, const float *a2, const uint16_t &size);
bool matrixCheck(float **m1, float **m2, const uint16_t &size);
void printArray(const float *arr, const uint16_t &size);
void printMatrix(float **arr, const uint16_t &size);

__global__ void matDotVec(float **arr, const uint16_t &size);

int main(int argc, char **argv){
    sanityCheck();
    return 0;
}



void sanityCheck()
{
    uint16_t n = BLK_SIZE;

    float **ptr = NULL;
    float testnxn[BLK_SIZE][BLK_SIZE];
    ptr = testnxn;
    float incr = 0.0;
    for (uint16_t i = 0; i < n; ++i){
        for (uint16_t j = 0; j < n; ++j){
            testnxn[i][j] = ++incr;
        }
    }
    float testnx1[n] = {10.0f, 11.0f, 12.0f};

    float *d_result;
    float **h_result;
    float *expected;
    uint32_t bytes = n * sizeof(float);
    uint32_t bytes_sqrd = bytes * bytes;

    h_result = (float**)malloc(bytes_sqrd);
    memset(h_result, 0.0f, bytes_sqrd);

    expected = (float*)malloc(bytes);
    expected[0] = 68.0f;
    expected[1] = 167.0f;
    expected[2] = 266.0f;
    

    hipMalloc((float**)&d_result, bytes_sqrd);
    hipMemcpy(d_result, testnxn, bytes_sqrd, hipMemcpyHostToDevice);
    // hipMemcpy(d_result, expected, bytes, hipMemcpyHostToDevice);

    // Call CUDA function here...
    matDotVec<<<1, n>>>(ptr, n);

    hipMemcpy(h_result, d_result, bytes_sqrd, hipMemcpyDeviceToHost);
    hipFree(d_result);

    if (matrixCheck(h_result, h_result, n)){
        std::cout << "Sanity check PASSED!" << std::endl;
    }
    else{
        std::cout << "Sanity check FAILED!" << std::endl;
        
    }
    printMatrix(h_result, n);
    free(h_result);
    free(expected);
}



bool arrayCheck(const float *a1, const float *a2, const uint16_t &size){
    for (uint16_t i = 0; i < size; ++i){
        if (a1[i] != a2[i]){            
            return false;
        }
    }
    return true;
}



bool matrixCheck(float **m1, float **m2, const uint16_t &size){
    for (uint16_t i = 0; i < size * size; ++i){
        if (m1[i] != m2[i]){            
            return false;
        }
    }
    return true;
}



void printArray(const float *arr, const uint16_t &size){
    for (uint16_t i = 0; i < size; ++i){
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}



void printMatrix(float **arr, const uint16_t &size){
    for (uint16_t i = 0; i < size; ++i){
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}


__global__ void matDotVec(float **arr, const uint16_t &size){
    
}