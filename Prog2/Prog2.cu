
#include <hip/hip_runtime.h>
#include <iostream>     // cout
#include <stdint.h>     // Standardized integers
#include <unistd.h>
#include <math.h>       // fabs
#include <time.h>       /* time */
#include <stdlib.h>     /* srand, rand */


const uint64_t BLK_SIZE = 2048; // 8192
const uint64_t SANITY_N = 3;    // Size of sanity check matrix
const float EPSILON = 0.0001;   // For sanity check: float == float

void testAdotB();
void testSlowAdotB();
void sanityCheck();
bool resultCheck(float *arr1, float *arr2);
void printArray(float *arr);

__global__ void matDotVec(float *m1, float *m2, float *arr, uint64_t size);
__global__ void matAddElementWise(float *m2, float *arr, uint64_t size);
__global__ void slowDotVec(float *m1, float *arr, float *res, uint64_t size);

int main(int argc, char **argv){
    // sanityCheck();
    srand (time(NULL));
    // testAdotB();
    testSlowAdotB();
    return 0;
}


void testAdotB(){
    uint64_t n = BLK_SIZE;
    uint64_t bytes = n * sizeof(float);
    uint64_t ptr_bytes = n * sizeof(float*);        // Just being safe
    uint64_t bytes_sqrd = bytes * bytes;

    float *h_matrix = (float*)malloc(ptr_bytes * ptr_bytes);   // To store "Matrix A"
    float *h_vec = (float*)malloc(bytes);           // To store "Vector B" and result

    float *d_matrix1;       // To store the data from host
    float *d_matrix2;       // To store the multiplication that happens
    float *d_vec;           // To store the data from host and result

    // Fill "Matrix A"
    for (uint64_t i = 0; i < n * n; ++i){
        h_matrix[i] = rand() % 1000;
    }

    // Fill "Vector B"
    for (uint64_t i = 0; i < n; ++i){    
        h_vec[i] = rand() % 1000;
    }

    // Allocate CUDA memory and copy data to device
    hipMalloc((float**)&d_matrix1, bytes_sqrd);
    hipMalloc((float**)&d_matrix2, bytes_sqrd);
    hipMalloc((float**)&d_vec, bytes);
    hipMemcpy(d_matrix1, h_matrix, bytes_sqrd, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

    int blockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, matDotVec, 0, n); 
    uint64_t gridSize = (n + blockSize - 1) / blockSize; 
    matDotVec<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_vec, n);
    hipDeviceSynchronize();    // Just in case
    matAddElementWise<<<gridSize, blockSize>>>(d_matrix2, d_vec, n);
    hipDeviceSynchronize();


    // Copy result back to host
    hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);


    // Free CUDA and host memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_vec);
    free(h_matrix);
    free(h_vec);
}


void testSlowAdotB(){
    uint64_t n = BLK_SIZE;
    uint64_t bytes = n * sizeof(float);
    uint64_t ptr_bytes = n * sizeof(float*);        // Just being safe
    uint64_t bytes_sqrd = bytes * bytes;

    float *h_matrix = (float*)malloc(ptr_bytes * ptr_bytes);   // To store "Matrix A"
    float *h_vec = (float*)malloc(bytes);           // To store "Vector B" and result

    float *d_matrix1;       // To store the data from host    
    float *d_vec;           // To store the data from host and result
    float *d_vec2;           // To store the data from host and result

    // Fill "Matrix A"
    for (uint64_t i = 0; i < n * n; ++i){    
        h_matrix[i] = rand() % 1000;
    }

    // Fill "Vector B"
    for (uint64_t i = 0; i < n; ++i){    
        h_vec[i] = rand() % 1000;
    }

    // Allocate CUDA memory and copy data to device
    hipMalloc((float**)&d_matrix1, bytes_sqrd);
    hipMalloc((float**)&d_vec, bytes);
    hipMalloc((float**)&d_vec2, bytes);
    hipMemcpy(d_matrix1, h_matrix, bytes_sqrd, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

    slowDotVec<<<1, 1>>>(d_matrix1, d_vec, d_vec2, n);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_vec, d_vec2, bytes, hipMemcpyDeviceToHost);


    // Free CUDA and host memory
    hipFree(d_matrix1);
    hipFree(d_vec);
    hipFree(d_vec2);
    free(h_matrix);
    free(h_vec);
}



void sanityCheck()
{
    uint64_t n = SANITY_N;
    uint64_t bytes = n * sizeof(float);
    uint64_t ptr_bytes = n * sizeof(float*);        // Just being safe
    uint64_t bytes_sqrd = bytes * bytes;

    float *h_matrix = (float*)malloc(ptr_bytes * ptr_bytes);   // To store "Matrix A"
    float *h_vec = (float*)malloc(bytes);           // To store "Vector B" and result
    float *expected = (float*)malloc(bytes);        // Sanity check array

    float *d_matrix1;       // To store the data from host
    float *d_matrix2;       // To store the multiplication that happens
    float *d_vec;           // To store the data from host and result
    float *d_vec2;          // To store the data from host and result

    float incr = 0.0;
    for (uint64_t i = 0; i < n * n; ++i){    
        h_matrix[i] = ++incr;
    }

    // "Vector B"
    h_vec[0] = 10.0f;
    h_vec[1] = 11.0f;
    h_vec[2] = 12.0f;
    
    // Expected values out of matrixCheck()
    expected[0] = 68.0f;
    expected[1] = 167.0f;
    expected[2] = 266.0f;
    

    // Allocate CUDA memory and copy data to device
    hipMalloc((float**)&d_matrix1, bytes_sqrd);
    hipMalloc((float**)&d_matrix2, bytes_sqrd);
    hipMalloc((float**)&d_vec, bytes);
    hipMalloc((float**)&d_vec2, bytes);
    hipMemcpy(d_matrix1, h_matrix, bytes_sqrd, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

    matDotVec<<<1, n * n>>>(d_matrix1, d_matrix2, d_vec, n);
    hipDeviceSynchronize();
    matAddElementWise<<<1, n>>>(d_matrix2, d_vec, n);
    hipDeviceSynchronize();

    // slow test:
    // slowDotVec<<<1, 1>>>(d_matrix1, d_vec, d_vec2, n);
    // cudaDeviceSynchronize();
    // cudaMemcpy(h_vec, d_vec2, bytes, cudaMemcpyDeviceToHost);


    // cudaFree(d_vec);
    // cudaFree(d_vec2);
    // cudaFree(d_matrix1);
    // cudaFree(d_matrix2);
    // end slow test


    // Copy result back to host & free memory
    hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_matrix1);
    hipFree(d_matrix2);

    if (resultCheck(expected, h_vec)){
        std::cout << "Sanity check PASSED!" << std::endl;
    }
    else{
        std::cout << "Sanity check FAILED!" << std::endl;
        
    }
    printArray(h_vec);

    // Free up device memory allocations
    free(h_matrix);
    free(h_vec);
    free(expected);
}



bool resultCheck(float *arr1, float *arr2){
    for (uint64_t i = 0; i < SANITY_N; ++i){
        if (fabs(arr1[i] - arr2[i]) > EPSILON){            
            return false;
        }
    }
    return true;
}




void printArray(float *arr){
    for (uint64_t i = 0; i < SANITY_N; ++i){
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}



__global__ void matDotVec(float *m1, float *m2, float *arr, uint64_t size){
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size * size){
        m2[idx] = m1[idx] * arr[idx % size];
    }
}

__global__ void matAddElementWise(float *m2, float *arr, uint64_t size){
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size * size){
        uint64_t arr_idx = idx % size;
        uint64_t mat_loc = arr_idx * size;
        uint64_t max = mat_loc + size;
        arr[arr_idx] = 0.0f;
        for (uint64_t i = mat_loc; i < max; ++i){
            arr[arr_idx] += m2[i];
        }
    }
}

__global__ void slowDotVec(float *m1, float *arr, float *res, uint64_t size){
    // Assumes this is ran on one thread
    uint64_t row = 0;
    for (uint64_t i = 0; i < size; ++i){
        res[i] = 0.0f;            
        for (uint64_t j = 0; j < size; ++j){
            res[i] += m1[row * size + j] * arr[j];
        }
        ++row;
    }
}