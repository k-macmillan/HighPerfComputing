
#include <hip/hip_runtime.h>
#include <iostream>     // cout
#include <stdint.h>     // Standardized integers


const uint16_t BLK_SIZE = 256;
void sanityCheck();
bool arrayCheck(const float *a1, const float *a2, const uint32_t &size);

int main(int argc, char **argv){
    sanityCheck();
    return 0;
}



void sanityCheck()
{
    const uint32_t n = 3;
    float testnxn[n][n];
    float incr = 0.0;
    for (uint8_t i = 0; i < n; ++i){
        for (uint8_t j = 0; j < n; ++j){
            testnxn[i][j] = ++incr;
        }
    }

    float testnx1[n] = {10.0f, 11.0f, 12.0f};
    float expected[n] = {68.0f, 167.0f, 266.0f};
    float result[n] = {0.0f, 0.0f, 0.0f};

    // Call CUDA function here...

    if (arrayCheck(expected, result, n)){
        std::cout << "Sanity check PASSED!" << std::endl;
    }
    else{
        std::cout << "Sanity check FAILED!" << std::endl;
    }
}



bool arrayCheck(const float *a1, const float *a2, const uint32_t &size){
    for (uint32_t i = 0; i < size; ++i){
        if (a1[i] != a2[i]){
            return false;
        }
    }
    return true;
}