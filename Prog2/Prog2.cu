/* File:        Prog2.cu
 *
 * Author:      Kyle MacMillan
 *
 * Assignment:  Programming Assignment #2
 *
 * Course:      CSC 792 - High Performance Computing
 *
 * Purpose:     Tests matrix multiplied with a vector and matrix added to a 
 *              matrix. This was done with a class function because there was a 
 *              lot of repeat code and I like compartmentalizing code. The 
 *              primary functionality of the class is overloaded to accept 
 *              arbitrary matrices/vectors, though they must be of size 
 *              specified at class instantiation. For example you can pass a 
 *              matrix and vector of your choosing to AdotBfast() and it will 
 *              skip the randomization of values.
 *
 *              To use this class simply declare it with the "n" you intend to 
 *              use for the n x n matrix and n x 1 vector. The class will 
 *              generate a random numbers to fill the matrix and vector. If you 
 *              want to pass it your own matrix and vector you can do so by 
 *              calling the appropriate function with the 1D matrix and vector 
 *              you would like to use. It was done this way because it was very 
 *              quick to do and greatly increases the usage range. 
 *
 *              All public functions, other than sanity check, return the 1D 
 *              array result, but is free'd on the next public function call. 
 *              The program intenionally allocates extra memory because I 
 *              wanted to keep the code clean. Extra memory allocation does not 
 *              hamper the performance testing because I did not include that in 
 *              the measurement.
 *
 *              When the class is destructed all allocated memory is freed on 
 *              both the host and device.
 *
 *              Users can specify a matrix/vector size or not, the program will 
 *              run regardless. I set a hard limit of matrix/vector size n being 
 *              1024. It could have been arbitrarily large and increased 
 *              complexity arbitrarily.
 *
 * Compile:     nvcc -o prog2 Prog2.cu -std=c++11
 *              make (with provided Makefil)
 *
 * Run:         ./prog2
 *              ./prog2 #
 *
 * Output:      Santiy check verfication output to screen
 *
 * Notes:       None, runs like a dream
 *
 */        



#include <hip/hip_runtime.h>
#include <iostream>     // cout
#include <stdint.h>     // uint64_t
#include <math.h>       // fabs
#include <time.h>       // time
#include <stdlib.h>     // srand, rand



/*------------------------------------------------------------------------------
 * Function:    fastDotVec
 * Purpose:     CUDA function. Stores matrix value multiplied by associated 
 *              array value into m2 for later use.
 *
 * In arg:      m1, m2, arr, size
 *
 * Out arg:     m2
 */
__global__ void fastDotVec(float *m1, float *m2, float *arr, uint64_t size){
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size * size){
        m2[idx] = m1[idx] * arr[idx % size];
    }
}



/*------------------------------------------------------------------------------
 * Function:    matAddElementWise
 * Purpose:     CUDA function. Walks N elements of m2, adding each to the 
 *              appropriate index in arr. This is the result of Matrix * Vector.
 *
 * In arg:      m2, arr, size
 *
 * Out arg:     arr
 */
__global__ void matAddElementWise(float *m2, float *arr, uint64_t size){
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size * size){
        uint64_t arr_idx = idx % size;
        uint64_t mat_loc = arr_idx * size;
        uint64_t max = mat_loc + size;
        arr[arr_idx] = 0.0f;
        for (uint64_t i = mat_loc; i < max; ++i){
            arr[arr_idx] += m2[i];
        }
    }
}



/*------------------------------------------------------------------------------
 * Function:    slowDotVec
 * Purpose:     CUDA function. This is to demonstrate running Matrix * Vector on 
 *              a single thread in a CUDA call so we can compare p = 1 to p = n. 
 *              The result is stored in res.
 *
 * In arg:      m1, arr, res, size
 *
 * Out arg:     res
 */
__global__ void slowDotVec(float *m1, float *arr, float *res, uint64_t size){
    // Assumes this is ran on one thread
    uint64_t row = 0;
    for (uint64_t i = 0; i < size; ++i){
        res[i] = 0.0f;
        for (uint64_t j = 0; j < size; ++j){
            res[i] += m1[row * size + j] * arr[j];
        }
        ++row;
    }
}



/*------------------------------------------------------------------------------
 * Function:    addMatrixFast
 * Purpose:     CUDA function. This is to demonstrate running Matrix + Matrix in 
 *              parallel on a CUDA-capabale card. Simply uses the "location" to 
 *              determine matrix index and add the values together. Result is 
 *              stored in m1.
 *
 * In arg:      m1, m2, size
 *
 * Out arg:     m1
 */
__global__ void addMatrixFast(float *m1, float *m2, uint64_t size){
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size){
        m1[idx] = m1[idx] + m2[idx];
    }
}



/*------------------------------------------------------------------------------
 * Function:    addMatrixSlow
 * Purpose:     CUDA function. This is to demonstrate running Matrix + Matrix on 
 *              a single thread in a CUDA device. Walks the entire array, adding 
 *              element-wise. Result is stored in m1.
 *
 * In arg:      m1, m2, size
 *
 * Out arg:     m1
 */
__global__ void addMatrixSlow(float *m1, float *m2, uint64_t size){
    for (uint64_t i = 0; i < size; ++i){
        m1[i] = m1[i] + m2[i];
    }
}



/*<! Class for the assignment */
class MatVec{
public:
    /*--------------------------------------------------------------------------
     * Function:    Constructor
     * Purpose:     Constructs the class, sets constant variables and 
     *              initializes srand.
     *
     * In arg:      size
     */
    MatVec(uint64_t size) : n(size){
        srand (time(NULL));
        bytes = n * sizeof(float);
        ptr_bytes = n * sizeof(float*);        // Just being safe
        bytes_sqrd = bytes * bytes;
    }



    /*--------------------------------------------------------------------------
     * Function:    Destructor
     * Purpose:     Cleans up/frees allocated memory.
     *
     */
    ~MatVec(){
        freeAll();
    }



    /*--------------------------------------------------------------------------
     * Function:    sanityCheckFast
     * Purpose:     Performs a sanity check on the fastDotVec and 
     *              matAddElementWise CUDA functions. Outputs pass or fail to 
     *              console.
     *
     */
    void sanityCheckFast(){
        allocateAll();
        expected = allocateMemory(bytes);
        // Set up identity matrix, sequential/expected vectors
        for (uint64_t i = 0; i < n; ++i){
            for (uint64_t j = 0; j < n; ++j){
                uint64_t pos = i * n + j;
                if (i == j){
                    h_matrix1[pos] = 1.0f;
                }
                else{
                    h_matrix1[pos] = 0.0f;
                }
            }
            h_vec[i] = i;
            expected[i] = i;
        }

        // Copy from host to device
        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

        // Perform calculations on device
        fastDotVec<<<n, n>>>(d_matrix1, d_matrix2, d_vec, n);
        hipDeviceSynchronize();
        matAddElementWise<<<1, n>>>(d_matrix2, d_vec, n);
        hipDeviceSynchronize();

        // Copy result back to host & free memory
        hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);

        if (resultCheck(expected, h_vec)){
            std::cout << "Fast sanity check PASSED!" << std::endl;
        }
        else{
            std::cout << "Fast sanity check FAILED!" << std::endl;
            
        }
        // printArray(h_vec);
        freeAll();
    }



    /*--------------------------------------------------------------------------
     * Function:    sanityCheckSlow
     * Purpose:     Performs a sanity check on the slowDotVec CUDA function. 
     *              Outputs pass or fail to console.
     *
     */
    void sanityCheckSlow(){
        freeAll();
        allocateAll();
        expected = allocateMemory(bytes);
        // Set up identity matrix, sequential/expected vectors
        for (uint64_t i = 0; i < n; ++i){
            for (uint64_t j = 0; j < n; ++j){
                uint64_t pos = i * n + j;
                if (i == j){
                    h_matrix1[pos] = 1.0f;
                }
                else{
                    h_matrix1[pos] = 0.0f;
                }
            }
            h_vec[i] = i;
            expected[i] = i;
        }


        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

        slowDotVec<<<1, 1>>>(d_matrix1, d_vec, d_vec2, n);
        hipDeviceSynchronize();
        hipMemcpy(h_vec, d_vec2, bytes, hipMemcpyDeviceToHost);

        if (resultCheck(expected, h_vec)){
            std::cout << "Slow sanity check PASSED!" << std::endl;
        }
        else{
            std::cout << "Slow sanity check FAILED!" << std::endl;
            
        }
        // printArray(h_vec);
    }



    /*--------------------------------------------------------------------------
     * Function:    AdotBfast
     * Purpose:     Frees memory, then allocates memory because we don't know 
     *              the previous class state. Then fills the matrix and array 
     *              with helper function fillMatrixArray(). Copies data to the 
     *              device via CudaMemcpy, performs the CUDA functions, 
     *              synchronizes, then transfers the data back to the host. 
     *              Returns the result array.
     *
     * In arg:      m, arr
     *
     * return:      h_vec
     */
    float* AdotBfast(float *m = NULL, float *arr = NULL){
        freeAll();
        allocateAll();
        
        fillMatrixArray(m, arr);

        // Copy host to device
        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

        // Perform operation
        fastDotVec<<<n, n>>>(d_matrix1, d_matrix2, d_vec, n);
        hipDeviceSynchronize();    // Just in case
        matAddElementWise<<<1, n>>>(d_matrix2, d_vec, n);
        hipDeviceSynchronize();


        // Copy result back to host
        hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);

        return h_vec;
        
    }



    /*--------------------------------------------------------------------------
     * Function:    AdotBslow
     * Purpose:     Frees memory, then allocates memory because we don't know 
     *              the previous class state. Then fills the matrix and array 
     *              with helper function fillMatrixArray(). Copies data to the 
     *              device via CudaMemcpy, performs the CUDA functions, 
     *              synchronizes, then transfers the data back to the host. 
     *              Returns the result array.
     *
     * In arg:      m, arr
     *
     * return:      h_vec
     */
    float* AdotBslow(float *m = NULL, float *arr = NULL){
        freeAll();
        allocateAll();

        fillMatrixArray(m, arr);

        // Copy host to device
        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);

        // Perform operation
        slowDotVec<<<1, 1>>>(d_matrix1, d_vec, d_vec2, n);
        hipDeviceSynchronize();

        // Copy result back to host
        hipMemcpy(h_vec, d_vec2, bytes, hipMemcpyDeviceToHost);

        return h_vec;
    }



    /*--------------------------------------------------------------------------
     * Function:    AplusBfast
     * Purpose:     Frees memory, then allocates memory because we don't know 
     *              the previous class state. Then fills both matrices with 
     *              helper function fillMatrixArray(). Copies data to the device 
     *              via CudaMemcpy, performs the CUDA function, synchronizes, 
     *              then transfers the data back to the host. Returns the result 
     *              array.
     *
     * In arg:      m1, m2
     *
     * return:      h_matrix1
     */
    float* AplusBfast(float *m1 = NULL, float *m2 = NULL){
        freeAll();
        allocateAll();
        
        fillMatrixMatrix(m1, m2);

        // Copy host to device
        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, h_matrix2, bytes_sqrd, hipMemcpyHostToDevice);

        // Perform operation
        addMatrixFast<<<n, n>>>(d_matrix1, d_matrix2, n * n);
        hipDeviceSynchronize();


        // Copy result back to host
        hipMemcpy(h_matrix1, d_matrix1, bytes_sqrd, hipMemcpyDeviceToHost);

        return h_matrix1;
    }



    /*--------------------------------------------------------------------------
     * Function:    AplusBslow
     * Purpose:     Frees memory, then allocates memory because we don't know 
     *              the previous class state. Then fills both matrices with 
     *              helper function fillMatrixArray(). Copies data to the device 
     *              via CudaMemcpy, performs the CUDA function, synchronizes, 
     *              then transfers the data back to the host. Returns the result 
     *              array.
     *
     * In arg:      m1, m2
     *
     * return:      h_matrix1
     */
    float* AplusBslow(float *m1 = NULL, float *m2 = NULL){
        freeAll();
        allocateAll();
        
        fillMatrixMatrix(m1, m2);

        // Copy host to device
        hipMemcpy(d_matrix1, h_matrix1, bytes_sqrd, hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, h_matrix2, bytes_sqrd, hipMemcpyHostToDevice);

        // Perform operation
        addMatrixSlow<<<1, 1>>>(d_matrix1, d_matrix2, n * n);
        hipDeviceSynchronize();


        // Copy result back to host
        hipMemcpy(h_matrix1, d_matrix1, bytes_sqrd, hipMemcpyDeviceToHost);

        return h_matrix1;
    }



    /*--------------------------------------------------------------------------
     * Function:    resultCheck
     * Purpose:     Performs checks between two arrays of size n. Verifies they 
     *              are the same to within an epsilon.
     *
     * In arg:      arr1, arr2
     *
     * return:      bool
     */
    bool resultCheck(float *arr1, float *arr2){
        for (uint64_t i = 0; i < n; ++i){
            if (fabs(arr1[i] - arr2[i]) > EPSILON){            
                return false;
            }
        }
        return true;
    }



    /*--------------------------------------------------------------------------
     * Function:    printArray
     * Purpose:     Prints an array of size n.
     *
     * In arg:      arr
     */
    void printArray(float *arr){
        for (uint64_t i = 0; i < n; ++i){
            std::cout << arr[i] << " ";
        }
        std::cout << std::endl;
    }

private:
    const float EPSILON = 0.0001;   // For sanity check
    const uint64_t n;               // Size of matrix (n x n)
    uint64_t bytes;                 // Bytes needed for array mallocs
    uint64_t ptr_bytes;             // Pointer bytes needed for mallocs
    uint64_t bytes_sqrd;            // Bytes for matrix mallocs

    // Dynamic memory
    float *h_matrix1;       // To store "Matrix A" and result matrix
    float *h_matrix2;       // To store "Matrix B"
    float *h_vec;           // To store "Vector B" and result

    float *d_matrix1;       // To store the data from host
    float *d_matrix2;       // To store the multiplication that happens
    float *d_vec;           // To store the data from host and result
    float *d_vec2;          // To store the data from host and result
    float *expected;        // To store expected sanity check result



    /*--------------------------------------------------------------------------
     * Function:    allocateMemory
     * Purpose:     Allocates memory on the host. Returns a pointer to the 
     *              allocated memory.
     *
     * return:      float pointer
     */
    float* allocateMemory(uint64_t size){
        return (float*)malloc(size);
    }



    /*--------------------------------------------------------------------------
     * Function:    allocateCudaMemory
     * Purpose:     Allocates memory on the device.
     *
     * return:      float pointer
     */
    void allocateCudaMemory(float *arr, uint64_t size){
        hipMalloc((float**)&arr, size);
    }



    /*--------------------------------------------------------------------------
     * Function:    allocateAll
     * Purpose:     Allocates all memory necessary for any function call in this 
     *              class.
     *
     */
    void allocateAll(){
        // Initialize all dynamic memory
        h_matrix1 = allocateMemory(ptr_bytes * ptr_bytes);
        h_matrix2 = allocateMemory(ptr_bytes * ptr_bytes);
        h_vec = allocateMemory(bytes);

        allocateCudaMemory(d_matrix1, bytes_sqrd);
        allocateCudaMemory(d_matrix2, bytes_sqrd);
        allocateCudaMemory(d_vec, bytes);
        allocateCudaMemory(d_vec2, bytes);
    }



    /*--------------------------------------------------------------------------
     * Function:    freeAll
     * Purpose:     Frees all memory allocated, if it is allocated. Sets 
     *              pointers to NULL so we know their state.
     *
     */
    void freeAll(){
        if (h_vec != NULL){
            delete h_vec;
            h_vec = NULL;
        }
        if (h_matrix1 != NULL){
            delete h_matrix1;
            h_matrix1 = NULL;
        }
        if (h_matrix2 != NULL){
            delete h_matrix2;
            h_matrix2 = NULL;
        }
        if (d_matrix1 != NULL){
            hipFree(d_matrix1);
            d_matrix1 = NULL;
        }
        if (d_matrix2 != NULL){
            hipFree(d_matrix2);
            d_matrix2 = NULL;
        }
        if (d_vec != NULL){
            hipFree(d_vec);
            d_vec = NULL;
        }
        if (d_vec2 != NULL){
            hipFree(d_vec2);
            d_vec2 = NULL;
        }
    }



    /*--------------------------------------------------------------------------
     * Function:    fillMatrixArray
     * Purpose:     Helper class that fills h_matrix1 and h_vec with random 
     *              numbers if there is no matrix AND array passed to this 
     *              function. Otherwise, it will assign m1 and arr to h_matrix1 
     *              and h_vec. The user must clear up any allocated memory on 
     *              their end.
     *
     * In arg:      m, arr
     */
    void fillMatrixArray(float *m = NULL, float *arr = NULL){
        if (m == NULL || arr == NULL){
            // Fill "Matrix A"
            for (uint64_t i = 0; i < n * n; ++i){
                h_matrix1[i] = rand() % 1000;
            }

            // Fill "Vector B"
            for (uint64_t i = 0; i < n; ++i){    
                h_vec[i] = rand() % 1000;
            }
        }
        else{
            h_matrix1 = m;
            h_vec = arr;
        }
    }



    /*--------------------------------------------------------------------------
     * Function:    fillMatrixMatrix
     * Purpose:     Helper class that fills h_matrix1 and h_matrix2 with random 
     *              numbers if there is no matrix1 AND matrix2 passed to this 
     *              function. Otherwise, it will assign m1 and m2 to h_matrix1 
     *              and h_matrix2. The user must clear up any allocated memory 
     *              on their end.
     *
     * In arg:      m, arr
     */
    void fillMatrixMatrix(float *m1 = NULL, float *m2 = NULL){
        if (m1 == NULL || m2 == NULL){
            // Fill "Matrix A"
            for (uint64_t i = 0; i < n * n; ++i){
                h_matrix1[i] = rand() % 1000;
            }

            // Fill "Matrix B"
            for (uint64_t i = 0; i < n * n; ++i){
                h_matrix2[i] = rand() % 1000;
            }
        }
        else{
            h_matrix1 = m1;
            h_matrix2 = m2;
        }
    }
};



/*------------------------------------------------------------------------------
     * Function:    main
     * Purpose:     Runs the necessary functions to satisfy programing 
     *              assignment #2. Checks to see if the user specified a random 
     *              matrix/vector size to start, otherwise runs with the default 
     *              size of 1024. Instantiates the MatVec class and calls all 
     *              functions necessary to demonstrate assignment completion.
     *
     * In arg:      argc, argv
     *
     * returns:     0, 1
     */
int main(int argc, char **argv){
    uint64_t n = 1024;
    if (argc > 1){
        try{
            n = atoll(argv[1]);
            if (n == 0){
                std::cout << "Invalid n. Applied default 1024" << std::endl;
                n = 1024;
            }
            if (n > 1024){
                std::cout << "Program quit. n must be less than 1025" << std::endl;
                return 1;
            }
        }
        catch(...){
            std::cout << "Program quit. Usage: \"prog2 #\"" << std::endl;
            return 1;
        }
    }

    // Test MatVec functionality
    MatVec test = MatVec(n);

    // Perform sanity checks
    test.sanityCheckFast();
    test.sanityCheckSlow();
    
    // Undergrad section
    test.AdotBfast();
    test.AdotBslow();

    // Grad section
    test.AplusBfast();
    test.AplusBslow();

    return 0;
}