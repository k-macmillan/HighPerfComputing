#include "hip/hip_runtime.h"
#include <iostream>     // cout
#include <stdint.h>     // Standardized integers
#include <unistd.h>


const uint16_t BLK_SIZE = 256;
void sanityCheck();
bool arrayCheck(const float *a1, const float *a2, const uint32_t &size);
void printArray(const float *a1, const uint32_t &size);

__global__ void matDotVec(const float *a1, const uint32_t &size);

int main(int argc, char **argv){
    sanityCheck();
    return 0;
}



void sanityCheck()
{
    const uint32_t n = 3;
    float testnxn[n][n];
    float incr = 0.0;
    for (uint8_t i = 0; i < n; ++i){
        for (uint8_t j = 0; j < n; ++j){
            testnxn[i][j] = ++incr;
        }
    }
    float testnx1[n] = {10.0f, 11.0f, 12.0f};

    float *d_result;
    float *h_result;
    float *expected;
    uint32_t bytes = n * sizeof(float);

    h_result = (float*)malloc(bytes);
    memset(h_result, 0.0f, bytes);

    expected = (float*)malloc(bytes);
    expected[0] = 68.0f;
    expected[1] = 167.0f;
    expected[2] = 266.0f;
    

    hipMalloc((float**)&d_result, bytes);
    hipMemcpy(d_result, h_result, bytes, hipMemcpyHostToDevice);

    // Call CUDA function here...
    matDotVec<<<1, n>>>(n);

    hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);
    hipFree(d_result);

    if (arrayCheck(expected, h_result, n)){
        std::cout << "Sanity check PASSED!" << std::endl;
    }
    else{
        std::cout << "Sanity check FAILED!" << std::endl;
        printArray(h_result, n);
    }
    
    free(h_result);
    free(expected);
}



bool arrayCheck(const float *a1, const float *a2, const uint32_t &size){
    for (uint32_t i = 0; i < size; ++i){
        if (a1[i] != a2[i]){            
            return false;
        }
    }
    return true;
}



void printArray(const float *a1, const uint32_t &size){
    for (uint32_t i = 0; i < 3; ++i){
        std::cout << a1[i] << " ";
    }
    std::cout << std::endl;
}



__global__ void matDotVec(const float *a1, const uint32_t &size){
    __syncthreads();
}